#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <png.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "include/util.h"

// Error checking macro for CUDA calls
#define CHECK_CUDA_ERROR(call)                                                                                 \
    do                                                                                                         \
    {                                                                                                          \
        hipError_t err = call;                                                                                \
        if (err != hipSuccess)                                                                                \
        {                                                                                                      \
            fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                                                \
        }                                                                                                      \
    } while (0)

// Create Gaussian kernel and return it as a device pointer
__host__ float *create_gaussian_kernel(int radius, float sigma, float **d_kernel)
{
    int kernel_size = 2 * radius + 1;
    float *kernel = (float *)malloc(kernel_size * sizeof(float));
    float sum = 0.0f;

    // Fill kernel with Gaussian values
    for (int i = 0; i < kernel_size; i++)
    {
        int x = i - radius;
        kernel[i] = expf(-(x * x) / (2 * sigma * sigma));
        sum += kernel[i];
    }

    // Normalize kernel
    for (int i = 0; i < kernel_size; i++)
    {
        kernel[i] /= sum;
    }

    // Allocate and copy kernel to device
    CHECK_CUDA_ERROR(hipMalloc(d_kernel, kernel_size * sizeof(float)));
    CHECK_CUDA_ERROR(hipMemcpy(*d_kernel, kernel, kernel_size * sizeof(float), hipMemcpyHostToDevice));

    return kernel;
}

// CUDA kernel for horizontal blur pass
__global__ void horizontal_blur_kernel(
    unsigned char *d_input,
    unsigned char *d_output,
    int width,
    int height,
    int radius,
    float *d_kernel)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        float r = 0.0f, g = 0.0f, b = 0.0f, a = 0.0f;
        const int kernel_size = 2 * radius + 1;

        for (int i = -radius; i <= radius; i++)
        {
            int ix = x + i;
            // Handle boundary conditions
            if (ix < 0)
                ix = 0;
            if (ix >= width)
                ix = width - 1;

            const int input_idx = (y * width + ix) * 4;
            const float weight = d_kernel[i + radius];

            r += d_input[input_idx + 0] * weight;
            g += d_input[input_idx + 1] * weight;
            b += d_input[input_idx + 2] * weight;
            a += d_input[input_idx + 3] * weight;
        }

        const int output_idx = (y * width + x) * 4;
        d_output[output_idx + 0] = (unsigned char)r;
        d_output[output_idx + 1] = (unsigned char)g;
        d_output[output_idx + 2] = (unsigned char)b;
        d_output[output_idx + 3] = (unsigned char)a;
    }
}

// CUDA kernel for vertical blur pass
__global__ void vertical_blur_kernel(
    unsigned char *d_input,
    unsigned char *d_output,
    int width,
    int height,
    int radius,
    float *d_kernel)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        float r = 0.0f, g = 0.0f, b = 0.0f, a = 0.0f;
        const int kernel_size = 2 * radius + 1;

        for (int i = -radius; i <= radius; i++)
        {
            int iy = y + i;
            // Handle boundary conditions
            if (iy < 0)
                iy = 0;
            if (iy >= height)
                iy = height - 1;

            const int input_idx = (iy * width + x) * 4;
            const float weight = d_kernel[i + radius];

            r += d_input[input_idx + 0] * weight;
            g += d_input[input_idx + 1] * weight;
            b += d_input[input_idx + 2] * weight;
            a += d_input[input_idx + 3] * weight;
        }

        const int output_idx = (y * width + x) * 4;
        d_output[output_idx + 0] = (unsigned char)r;
        d_output[output_idx + 1] = (unsigned char)g;
        d_output[output_idx + 2] = (unsigned char)b;
        d_output[output_idx + 3] = (unsigned char)a;
    }
}

// Host function to apply Gaussian blur using CUDA
void apply_gaussian_blur_cuda(png_bytep *row_pointers, int width, int height, int radius)
{
    const size_t image_size = width * height * 4 * sizeof(unsigned char);
    unsigned char *h_image = (unsigned char *)malloc(image_size);

    // Copy row_pointers to a contiguous memory block
    for (int y = 0; y < height; y++)
    {
        memcpy(&h_image[y * width * 4], row_pointers[y], width * 4);
    }

    // Calculate sigma based on radius
    float sigma = radius / 2.0f;

    // Create and copy Gaussian kernel to device
    float *d_kernel;
    float *h_kernel = create_gaussian_kernel(radius, sigma, &d_kernel);

    // Allocate device memory
    unsigned char *d_input, *d_output, *d_temp;
    CHECK_CUDA_ERROR(hipMalloc(&d_input, image_size));
    CHECK_CUDA_ERROR(hipMalloc(&d_output, image_size));
    CHECK_CUDA_ERROR(hipMalloc(&d_temp, image_size));

    // Copy image data to device
    CHECK_CUDA_ERROR(hipMemcpy(d_input, h_image, image_size, hipMemcpyHostToDevice));

    // Define grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x,
                 (height + blockDim.y - 1) / blockDim.y);

    // Apply horizontal blur
    horizontal_blur_kernel<<<gridDim, blockDim>>>(d_input, d_temp, width, height, radius, d_kernel);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // Apply vertical blur
    vertical_blur_kernel<<<gridDim, blockDim>>>(d_temp, d_output, width, height, radius, d_kernel);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // Copy result back to host
    CHECK_CUDA_ERROR(hipMemcpy(h_image, d_output, image_size, hipMemcpyDeviceToHost));

    // Copy back to row_pointers format
    for (int y = 0; y < height; y++)
    {
        memcpy(row_pointers[y], &h_image[y * width * 4], width * 4);
    }

    // Clean up
    free(h_image);
    free(h_kernel);
    CHECK_CUDA_ERROR(hipFree(d_input));
    CHECK_CUDA_ERROR(hipFree(d_output));
    CHECK_CUDA_ERROR(hipFree(d_temp));
    CHECK_CUDA_ERROR(hipFree(d_kernel));
}

int main(int argc, char *argv[])
{
    const char *input_file = "image.png";
    const char *output_file = "cuda_out.png";
    int blur_radius = 10; // Default value

    if (argc > 1)
    {
        blur_radius = atoi(argv[1]);
        if (blur_radius <= 0)
        {
            printf("Invalid blur radius. Using default value: 10\n");
            blur_radius = 10;
        }
        input_file = argv[2];
    }
    else
    {
        printf("No blur radius specified. Using default value: 10\n");
    }
    printf("Using blur radius: %d\n", blur_radius);

    png_bytep *row_pointers;
    int width, height;
    clock_t start, end, read_start, read_end, write_start, write_end;
    double cpu_time_used, read_time_used, write_time_used;

    // Start measuring read time
    printf("Reading image from %s\n", input_file);
    read_start = clock();
    read_png_file(input_file, &row_pointers, &width, &height);
    read_end = clock();
    read_time_used = ((double)(read_end - read_start)) / CLOCKS_PER_SEC;
    printf("Image read successfully\nImage dimensions: %d x %d\n\n", width, height);

    // Start measuring processing time
    printf("Starting CUDA Blurring Process\n");
    start = clock();
    apply_gaussian_blur_cuda(row_pointers, width, height, blur_radius);
    end = clock();
    cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
    printf("CUDA Blurring Process Completed\n\n");

    // Start measuring write time
    printf("Writing image to %s\n", output_file);
    write_start = clock();
    write_png_file(output_file, row_pointers, width, height);
    write_end = clock();
    write_time_used = ((double)(write_end - write_start)) / CLOCKS_PER_SEC;
    printf("Image written successfully\n\n");

    printf("Freeing memory\n");
    for (int y = 0; y < height; y++)
    {
        free(row_pointers[y]);
    }
    free(row_pointers);
    printf("Memory freed\n\n");

    printf("Execution Summary:\n");
    printf("Time taken for reading: %f seconds\n", read_time_used);
    printf("Time taken for CUDA Gaussian blur with %d radius: %f seconds\n", blur_radius, cpu_time_used);
    printf("Time taken for writing: %f seconds\n", write_time_used);

    return 0;
}
